﻿#include "DataLoader.hpp"
#include "CNN.hpp"
#include "MLP.hpp"

void fashion_cnn() {

    auto TRAIN_FASHION = DataLoader::cnn_load_one_channel("fashion", "train");
    auto TEST_FASHION = DataLoader::cnn_load_one_channel("fashion", "test");

    auto& train_labels_data = std::get<0>(TRAIN_FASHION);
    auto& train_images_data = std::get<1>(TRAIN_FASHION);

    auto& test_labels_data = std::get<0>(TEST_FASHION);
    auto& test_images_data = std::get<1>(TEST_FASHION);
 
    train_labels_data.erase(train_labels_data.begin(), train_labels_data.begin() + 59000);
    train_images_data.erase(train_images_data.begin(), train_images_data.begin() + 59000);

    test_labels_data.erase(test_labels_data.begin(), test_labels_data.begin() + 9500);
    test_images_data.erase(test_images_data.begin(), test_images_data.begin() + 9500);


    ConvLayer::ConvLayerHyperparameters conv_params1;
    conv_params1.padding = true;
    conv_params1.pooling = true;
    conv_params1.activation = true;
    conv_params1.debug = false;
    conv_params1.learning_rate = 0.002;
    conv_params1.padding_size = 1;
    conv_params1.stride = 1;
    conv_params1.pool_size = 2;
    conv_params1.activation_func = Activation::RELU;
    conv_params1.init = Initializer::N_HE;
    conv_params1.pool_mode = PoolMode::MAX;

    ConvLayer::ConvLayerHyperparameters conv_params2;
    conv_params2.padding = true;
    conv_params2.pooling = true;
    conv_params2.activation = true;
    conv_params2.debug = false;
    conv_params2.learning_rate = 0.002;
    conv_params2.padding_size = 1;
    conv_params2.stride = 1;
    conv_params2.pool_size = 2;
    conv_params2.activation_func = Activation::RELU;
    conv_params2.init = Initializer::N_HE;
    conv_params2.pool_mode = PoolMode::MAX;

    std::vector<ConvLayer> conv_layers;
    conv_layers.emplace_back(1, 16, 3, conv_params1);
    conv_layers.emplace_back(16, 4, 3, conv_params2);

    DenseLayer::DenseLayerHyperparameters dense_params;
    dense_params.learning_rate = 0.002;
    dense_params.activation_func = Activation::RELU;
    dense_params.use_bias = true;
    dense_params.init = Initializer::N_HE;

    auto first_dense_layer = 8 * 8 * 4;

    std::vector<DenseLayer> dense_layers;
    dense_layers.emplace_back(first_dense_layer, 16, dense_params);

    DenseLayer::DenseLayerHyperparameters output_params = dense_params;
    output_params.use_activation = false;
    dense_layers.emplace_back(16, 10, output_params);

    CNN cnn(conv_layers, dense_layers);

    CNN::CNNHyperparameters train_params;
    train_params.epochs = 1;
    train_params.batch_size = 32;
    train_params.learning_rate = 0.002;
    train_params.verbose = true;
    train_params.print_every = 1;
    train_params.save_measure = true;

    TrainingMetrics metrics = cnn.train(train_images_data, train_labels_data, test_images_data, test_labels_data, train_params);
}

void fashion_mlp() {

    auto TRAIN_FASHION = DataLoader::mlp_load_one_channel("fashion", "train");
    auto TEST_FASHION = DataLoader::mlp_load_one_channel("fashion", "test");

    auto& train_labels_data = std::get<0>(TRAIN_FASHION);
    auto& train_images_data = std::get<1>(TRAIN_FASHION);

    auto& test_labels_data = std::get<0>(TEST_FASHION);
    auto& test_images_data = std::get<1>(TEST_FASHION);

    train_labels_data.erase(train_labels_data.begin(), train_labels_data.begin() + 59000);
    train_images_data.erase(train_images_data.begin(), train_images_data.begin() + 59000);

    test_labels_data.erase(test_labels_data.begin(), test_labels_data.begin() + 9500);
    test_images_data.erase(test_images_data.begin(), test_images_data.begin() + 9500);
 
    MLP::MLPHyperparameters h;
    h.learning_rate = 0.002;
    h.batch = 32;
    h.epochs = 1;
    h.shuffle = true;
    h.save_measure = true;
    h.layers = { 784, 16, 10 };
    h.initializer = Initializer::N_HE;
    h.optimizer = Optimizer::NONE;
    h.activation_func = Activation::RELU;
    h.verbose = true;
    h.print_every = 1;

    MLP mlp(h);

    mlp.train(train_images_data, train_labels_data, test_images_data, test_labels_data);
}


int main() {

    fashion_cnn();

	return 0;
}
